/* File:     vec_add.cu
 * Purpose:  Implement vector addition on a gpu using cuda
 *
 * Compile:  nvcc [-g] [-G] -o vec_add vec_add.cu
 * Run:      ./vec_add
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

__global__ void Vec_add(unsigned int x[], unsigned int y[], unsigned int z[], int n) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < n){
        z[thread_id] = x[thread_id] + y[thread_id];
    }
}


int main(int argc, char* argv[]) {
    int n, m;
    unsigned int *h_x, *h_y, *h_z;
    unsigned int *d_x, *d_y, *d_z;
    size_t size;

    /* Define vector length */
    n = 2621440;
    m = 320;
    size = m * n * sizeof(unsigned int);

    // Allocate memory for the vectors on host memory.
    h_x = (unsigned int*) malloc(size);
    h_y = (unsigned int*) malloc(size);
    h_z = (unsigned int*) malloc(size);

    for (int i = 0; i < n * m; i++) {
        h_x[i] = i+1;
        h_y[i] = n-i;
    }

    printf("Input size = %d\n", n * m);

    // Print original vectors.
    /*printf("h_x = ");
    for (int i = 0; i < m; i++){
        printf("%u ", h_x[i]);
    }
    printf("\n\n");
    printf("h_y = ");
    for (int i = 0; i < m; i++){
        printf("%u ", h_y[i]);
    }
    printf("\n\n");*/

    // Event creation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time1 = 0;

    /* Allocate vectors in device memory */
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    
    // Start timer
    hipEventRecord( start, 0 );

    /* Kernel Call */
    Vec_add<<<(n * m) / 256, 256>>>(d_x, d_y, d_z, n * m);

    // End timer
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time1, start, stop );

    hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);
    /*printf("The sum is: \n");
    for (int i = 0; i < m; i++){
        printf("%u ", h_z[i]);
    }
    printf("\n");*/

    printf("Execution time = %f ms\n", time1);

    /* Free device memory */
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    /* Free host memory */
    free(h_x);
    free(h_y);
    free(h_z);

    return 0;
}  /* main */
