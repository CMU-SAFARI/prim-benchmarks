#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define THREAD 128

#define T int

__global__ void gemv(int m, int n, T *adim, T *b, T *d_ans);

void cgemv(int m, int n, T *adim, T *b, T *d_ans);

double gettime()
{
struct timeval tv;
gettimeofday(&tv, NULL);
return tv.tv_sec + (double)tv.tv_usec*1.0e-6;
}

int main(int argc, char **argv)
{
/* for CPU */
int i, j;
int *bdim, *c, *ans, *h_ans;
//double start, stop;
//double cpu_time, gpu_time;
int n = 8192;
int m = 20480;

bdim = (T*)malloc(sizeof(T) *m*n);
c = (T*)malloc(sizeof(T) *n);
ans = (T*)malloc(sizeof(T) *m);
h_ans = (T*)malloc(sizeof(T) *m);

/* for GPU */
T *d_bdim, *d_c, *d_ans;
hipMalloc((void **)&d_bdim, sizeof(T)*m*n);
hipMalloc((void **)&d_c, sizeof(T)*n);
hipMalloc((void **)&d_ans, sizeof(T)*m);

for(i = 0; i < n; i++)
{
c[i] = 1;
for(j = 0; j < m; j++)
bdim[i*m+j] = 1;
}

//start = gettime();
cgemv(m, n, bdim, c, ans);
//stop = gettime();
//cpu_time=stop - start;

// Event creation
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
float time1 = 0;


hipMemcpy(d_bdim, bdim, sizeof(T)*m*n, hipMemcpyHostToDevice);
hipMemcpy(d_c, c, sizeof(T)*n, hipMemcpyHostToDevice);

// Start timer
hipEventRecord( start, 0 );
//start = gettime();
gemv<<<m, THREAD>>>(m, n, d_bdim, d_c, d_ans);
//stop = gettime();
// End timer
hipEventRecord( stop, 0 );
hipEventSynchronize( stop );
hipEventElapsedTime( &time1, start, stop );

//gpu_time=stop - start;

hipMemcpy(h_ans, d_ans, sizeof(T)*m, hipMemcpyDeviceToHost);

//printf("cpu_time : %.6f[sec]\n",cpu_time);
//printf("gpu_time : %.6f[sec]\n",gpu_time);
//printf("%f x\n", cpu_time / gpu_time);


for(i = 0; i < m; i++)
printf("%d -- %d\n", ans[i], h_ans[i]);

printf("Execution time = %f ms\n", time1);


free(bdim);
free(c);
free(ans);
free(h_ans);
hipFree(d_bdim);
hipFree(d_c);
hipFree(d_ans);

return 0;
} 

__global__ void gemv(int m, int n, T* adim, T* b, T* d_ans)
{
int i;
int div = n/THREAD;
__shared__ T tmp[THREAD];

tmp[threadIdx.x] = 0.0;

for(i = 0; i < div; i++)
{
tmp[threadIdx.x] += adim[blockIdx.x*n+i*THREAD+threadIdx.x] * b[i * THREAD + threadIdx.x];
}
if(threadIdx.x < m%THREAD)
tmp[threadIdx.x] += adim[blockIdx.x*n+THREAD*div+threadIdx.x] * b[THREAD * div + threadIdx.x];

__syncthreads();

for(i = THREAD / 2; i > 31; i = i / 2)
{
if(threadIdx.x < i)
tmp[threadIdx.x] += tmp[threadIdx.x + i];
__syncthreads();
}

if(threadIdx.x < 16)
{
tmp[threadIdx.x] += tmp[threadIdx.x + 16];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 8];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 4];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 2];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 1];
__syncthreads();
}


if(threadIdx.x == 0)
d_ans[blockIdx.x] = tmp[0];

}

void cgemv(int m, int n, T *adim, T *b, T *d_ans)
{
int i, j;

for(i = 0; i < m; i++)
for(j = 0; j < n; j++)
d_ans[i] += adim[i*n+j] * b[j];

}
