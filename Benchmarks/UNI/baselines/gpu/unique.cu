/***************************************************************************
 *cr
 *cr            (C) Copyright 2015 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/*
  In-Place Data Sliding Algorithms for Many-Core Architectures, presented in ICPP’15

  Copyright (c) 2015 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Authors: Juan Gómez-Luna (el1goluj@uco.es, gomezlun@illinois.edu), Li-Wen Chang (lchang20@illinois.edu)
*/

#include "ds.h"
#include "kernel.cu"

// Sequential CPU version
void cpu_unique(T* output, T* input, int elements){
  int j = 0;
  output[j] = input[j];
  j++;
  for (int i = 1; i < elements; i++){
    if (input[i] != input[i-1]){
      output[j] = input[i];
      j++;		
    }
  }
}

int main(int argc, char **argv){

  // Syntax verification
  if (argc != 4) {
      printf("Wrong format\n");
      printf("Syntax: %s <Device Input (%% elements) numElements>\n",argv[0]);
      exit(1);
  }
  int device = atoi(argv[1]);
  int input = atoi(argv[2]);
  int numElements = atoi(argv[3]);
  size_t size = numElements * sizeof(T);

  // Set device
  hipDeviceProp_t device_properties;
  hipGetDeviceProperties(&device_properties,device);
  hipSetDevice(device);

  printf("DS Unique on %s\n", device_properties.name);
  printf("Thread block size = %d\n", L_DIM);
  printf("Coarsening factor = %d\n", REGS);
#ifdef FLOAT
  printf("Single precision array: %d elements\n", numElements);
#elif INT
  printf("Integer array: %d elements\n", numElements);
#else
  printf("Double precision array: %d elements\n", numElements);
#endif

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float time1 = 0;
  float time2 = 0;

  // Allocate the host input vector A
  T *h_A = (T*)malloc(size);

  // Allocate the host output vectors
  T *h_B = (T*)malloc(size);
  T *h_C = (T*)malloc(size);

  // Allocate the device input vector A
  T *d_A = NULL;
  hipMalloc((void **)&d_A, size);

#define WARMUP 0
#define REP 1
  int value1 = 0;
  int value2 = 1;
  int value3 = 2;
  int value4 = 3;
  unsigned int flagM = 0;
  for(int iteration = 0; iteration < REP+WARMUP; iteration++){
    // Initialize the host input vectors
    srand(2014);
    for(int i = 0; i < numElements; i++){
    	h_A[i] = value1;
        if(i >= numElements/4 && i < numElements/2) h_A[i] = value2;
        if(i >= numElements/2 && i < 3*numElements/4) h_A[i] = value3;
        if(i >= 3*numElements/4 && i < numElements) h_A[i] = value4;
    }
    int M = (numElements * input)/100;
    int m = M;
    while(m>0){
        int x = (int)(numElements*(((float)rand()/(float)RAND_MAX)));
        if(h_A[x]==value1 || h_A[x]==value2 || h_A[x]==value3 || h_A[x]==value4){
    	    h_A[x] = x+2;
            m--;
        }
    }

#if PRINT
    printf("\n");
    for(int i = 0; i < numElements; ++i){
        printf("%d ",*(h_A+i));
    }
    printf("\n");
#endif

    // Copy the host input vector A in host memory to the device input vector in device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int ldim = L_DIM;
    // Atomic flags
    unsigned int* d_flags = NULL;
    int num_flags = numElements % (ldim * REGS) == 0 ? numElements / (ldim * REGS) : numElements / (ldim * REGS) + 1;
    unsigned int *flags = (unsigned int *)calloc(sizeof(unsigned int), num_flags + 2);
    flags[0] = 1;
    flags[num_flags + 1] = 0;
    hipMalloc((void **)&d_flags, (num_flags + 2) * sizeof(unsigned int));
    hipMemcpy(d_flags, flags, (num_flags + 2) * sizeof(unsigned int), hipMemcpyHostToDevice);
    free(flags);
    // Number of work-groups/thread blocks
    int num_wg = num_flags;

    // Start timer
    hipEventRecord( start, 0 );

    // Kernel launch
    unique<<<num_wg, ldim>>>(d_A, d_A, numElements, d_flags);

    hipMemcpy(&flagM, d_flags + num_flags, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // End timer
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time1, start, stop );
    if(iteration >= WARMUP) time2 += time1;

    if(iteration == REP+WARMUP-1){
      float timer = time2 / REP;
      double bw = (double)((numElements + flagM) * sizeof(T)) / (double)(timer * 1000000.0);
      printf("Execution time = %f ms, Throughput = %f GB/s\n", timer, bw);
    }

    // Free flags
    hipFree(d_flags);
  }
  // Copy to host memory
  hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost);

  // CPU execution for comparison
  cpu_unique(h_C, h_A, numElements);

  // Verify that the result vector is correct
#if PRINT
  for(int i = 0; i < numElements; ++i){
     printf("%d ",*(h_B+i));
  }
  printf("\n");
  for(int i = 0; i < numElements; ++i){
      printf("%d ",*(h_C+i));
  }
  printf("\n");
#endif
  for (int i = 0; i < flagM - 1; ++i){
      if (h_B[i] != h_C[i]){
          fprintf(stderr, "Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
      }
  }
  printf("Test PASSED\n");

  // Free device global memory
  hipFree(d_A);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
