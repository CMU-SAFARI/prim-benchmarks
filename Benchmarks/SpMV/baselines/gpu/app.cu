#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <stdint.h>

#include "../../support/matrix.h"
#include "../../support/params.h"
#include "../../support/timer.h"
#include "../../support/utils.h"

__global__ void spmv_kernel(CSRMatrix csrMatrix, float* inVector, float* outVector) {
    unsigned int row = blockIdx.x*blockDim.x + threadIdx.x;
    if(row < csrMatrix.numRows) {
        float sum = 0.0f;
        for(unsigned int i = csrMatrix.rowPtrs[row]; i < csrMatrix.rowPtrs[row + 1]; ++i) {
            struct Nonzero nonzero = csrMatrix.nonzeros[i];
            sum += inVector[nonzero.col]*nonzero.value;
        }
        outVector[row] = sum;
    }
}

int main(int argc, char** argv) {

    // Process parameters
    struct Params p = input_params(argc, argv);

    // Initialize SpMV data structures
    PRINT_INFO(p.verbosity >= 1, "Reading matrix %s", p.fileName);
    struct COOMatrix cooMatrix = readCOOMatrix(p.fileName);
    PRINT_INFO(p.verbosity >= 1, "    %u rows, %u columns, %u nonzeros", cooMatrix.numRows, cooMatrix.numCols, cooMatrix.numNonzeros);
    struct CSRMatrix csrMatrix = coo2csr(cooMatrix);
    float* inVector = (float*) malloc(csrMatrix.numCols*sizeof(float));
    float* outVector = (float*) malloc(csrMatrix.numRows*sizeof(float));
    initVector(inVector, csrMatrix.numCols);

    // Allocate data structures on GPU
    CSRMatrix csrMatrix_d;
    csrMatrix_d.numRows = csrMatrix.numRows;
    csrMatrix_d.numCols = csrMatrix.numCols;
    csrMatrix_d.numNonzeros = csrMatrix.numNonzeros;
    hipMalloc((void**) &csrMatrix_d.rowPtrs, (csrMatrix_d.numRows + 1)*sizeof(unsigned int));
    hipMalloc((void**) &csrMatrix_d.nonzeros, csrMatrix_d.numNonzeros*sizeof(struct Nonzero));
    float* inVector_d;
    hipMalloc((void**) &inVector_d, csrMatrix_d.numCols*sizeof(float));
    float* outVector_d;
    hipMalloc((void**) &outVector_d, csrMatrix_d.numRows*sizeof(float));

    // Copy data to GPU
    hipMemcpy(csrMatrix_d.rowPtrs, csrMatrix.rowPtrs, (csrMatrix_d.numRows + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrMatrix_d.nonzeros, csrMatrix.nonzeros, csrMatrix_d.numNonzeros*sizeof(struct Nonzero), hipMemcpyHostToDevice);
    hipMemcpy(inVector_d, inVector, csrMatrix_d.numCols*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Calculating result on GPU
    PRINT_INFO(p.verbosity >= 1, "Calculating result on GPU");
    Timer timer;
    startTimer(&timer);
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (csrMatrix_d.numRows + numThreadsPerBlock - 1)/numThreadsPerBlock;
    spmv_kernel <<< numBlocks, numThreadsPerBlock >>> (csrMatrix_d, inVector_d, outVector_d);
    hipDeviceSynchronize();
    stopTimer(&timer);
    if(p.verbosity == 0) PRINT("%f", getElapsedTime(timer)*1e3);
    PRINT_INFO(p.verbosity >= 1, "    Elapsed time: %f ms", getElapsedTime(timer)*1e3);

    // Copy data from GPU
    hipMemcpy(outVector, outVector_d, csrMatrix_d.numRows*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Calculating result on CPU
    PRINT_INFO(p.verbosity >= 1, "Calculating result on CPU");
    float* outVectorReference = (float*) malloc(csrMatrix.numRows*sizeof(float));
    for(uint32_t rowIdx = 0; rowIdx < csrMatrix.numRows; ++rowIdx) {
        float sum = 0.0f;
        for(uint32_t i = csrMatrix.rowPtrs[rowIdx]; i < csrMatrix.rowPtrs[rowIdx + 1]; ++i) {
            uint32_t colIdx = csrMatrix.nonzeros[i].col;
            float value = csrMatrix.nonzeros[i].value;
            sum += inVector[colIdx]*value;
        }
        outVectorReference[rowIdx] = sum;
    }

    // Verify the result
    PRINT_INFO(p.verbosity >= 1, "Verifying the result");
    for(uint32_t rowIdx = 0; rowIdx < csrMatrix.numRows; ++rowIdx) {
        float diff = (outVectorReference[rowIdx] - outVector[rowIdx])/outVectorReference[rowIdx];
        const float tolerance = 0.00001;
        if(diff > tolerance || diff < -tolerance) {
            PRINT_ERROR("Mismatch at index %u (CPU result = %f, DPU result = %f)", rowIdx, outVectorReference[rowIdx], outVector[rowIdx]);
        }
    }

    // Deallocate data structures
    freeCOOMatrix(cooMatrix);
    freeCSRMatrix(csrMatrix);
    free(inVector);
    free(outVector);
    free(outVectorReference);

    return 0;

}

