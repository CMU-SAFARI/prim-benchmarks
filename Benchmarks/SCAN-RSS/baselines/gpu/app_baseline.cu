#include "hip/hip_runtime.h"
/*
* JGL@SAFARI
*/

/**
* GPU code with Thrust
*/
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <unistd.h>
#include <getopt.h>
#include <assert.h>

#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <math.h>
#include <sys/time.h>

#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/copy.h>

#include "../../support/common.h"
#include "../../support/timer.h"

#define ANSI_COLOR_RED     "\x1b[31m"
#define ANSI_COLOR_GREEN   "\x1b[32m"
#define ANSI_COLOR_RESET   "\x1b[0m"

// Pointer declaration
static T* A;
static T* C;
static T* C2;

/**
* @brief creates input arrays
* @param nr_elements how many elements in input arrays
*/
static void read_input(T* A, unsigned int nr_elements) {
    //srand(0);
    printf("nr_elements\t%u\t", nr_elements);
    for (unsigned int i = 0; i < nr_elements; i++) {
        //A[i] = (T) (rand()) % 2;
        A[i] = i;
    }
}

/**
* @brief compute output in the host
*/
static void scan_host(T* C, T* A, unsigned int nr_elements) {
    C[0] = A[0];
    for (unsigned int i = 1; i < nr_elements; i++) {
        C[i] = C[i - 1] + A[i - 1];
    }
}

// Params ---------------------------------------------------------------------
typedef struct Params {
    unsigned int   input_size;
    int   n_warmup;
    int   n_reps;
    int   exp;
    int   n_threads;
}Params;

void usage() {
    fprintf(stderr,
        "\nUsage:  ./program [options]"
        "\n"
        "\nGeneral options:"
        "\n    -h        help"
        "\n    -w <W>    # of untimed warmup iterations (default=1)"
        "\n    -e <E>    # of timed repetition iterations (default=3)"
        "\n    -x <X>    Weak (0) or strong (1) scaling (default=0)"
        "\n    -t <T>    # of threads (default=8)"
        "\n"
        "\nBenchmark-specific options:"
        "\n    -i <I>    input size (default=640 * 3932160 elements)"
        "\n");
}

struct Params input_params(int argc, char **argv) {
    struct Params p;
    p.input_size    = 1258291200;
    p.n_warmup      = 1;
    p.n_reps        = 3;
    p.exp           = 0;
    p.n_threads     = 8;

    int opt;
    while((opt = getopt(argc, argv, "hi:w:e:x:t:")) >= 0) {
        switch(opt) {
        case 'h':
        usage();
        exit(0);
        break;
        case 'i': p.input_size    = atoi(optarg); break;
        case 'w': p.n_warmup      = atoi(optarg); break;
        case 'e': p.n_reps        = atoi(optarg); break;
        case 'x': p.exp           = atoi(optarg); break;
        case 't': p.n_threads     = atoi(optarg); break;
        default:
            fprintf(stderr, "\nUnrecognized option!\n");
            usage();
            exit(0);
        }
    }
    assert(p.n_threads > 0 && "Invalid # of threads!");

    return p;
}

/**
* @brief Main of the Host Application.
*/
int main(int argc, char **argv) {

    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);
    hipSetDevice(0);

    struct Params p = input_params(argc, argv);

    unsigned int nr_of_dpus = 1;
    
    unsigned int i = 0;
    const unsigned int input_size = p.exp == 0 ? p.input_size * nr_of_dpus : p.input_size;

    // Input/output allocation
    A = (T*)malloc(input_size * sizeof(T));
    C = (T*)malloc(input_size * sizeof(T));
    C2 = (T*)malloc(input_size * sizeof(T));
    T *bufferA = A;
    T *bufferC = C2;

    // Create an input file with arbitrary data.
    read_input(A, input_size);

    // Timer declaration
    Timer timer;
    float time_gpu = 0;

    thrust::host_vector<T> h_output(input_size);

    // Loop over main kernel
    for(int rep = 0; rep < p.n_warmup + p.n_reps; rep++) {

        // Compute output on CPU (performance comparison and verification purposes)
        if(rep >= p.n_warmup)
            start(&timer, 0, rep - p.n_warmup);
        scan_host(C, A, input_size);
        if(rep >= p.n_warmup)
            stop(&timer, 0);


        // Event creation
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float time1 = 0;

        thrust::device_vector<T> d_input(input_size);
        hipMemcpy(thrust::raw_pointer_cast(&d_input[0]), A, input_size * sizeof(T), hipMemcpyHostToDevice);

        // Start timer
        hipEventRecord( start, 0 );
        thrust::exclusive_scan(d_input.begin(),d_input.end(),d_input.begin());
        // End timer
        hipEventRecord( stop, 0 );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &time1, start, stop );
        time_gpu += time1;

        h_output = d_input;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Print timing results
    printf("CPU ");
    print(&timer, 0, p.n_reps);
    printf("Kernel (ms):");
    printf("%f\n", time_gpu / p.n_reps);

    // Check output
    bool status = true;
    for (i = 0; i < input_size; i++) {
        if(C[i] != h_output[i]){ 
            status = false;
            printf("%d: %lu -- %lu\n", i, C[i], h_output[i]);
        }
    }
    if (status) {
        printf("[" ANSI_COLOR_GREEN "OK" ANSI_COLOR_RESET "] Outputs are equal\n");
    } else {
        printf("[" ANSI_COLOR_RED "ERROR" ANSI_COLOR_RESET "] Outputs differ!\n");
    }

    // Deallocation
    free(A);
    free(C);
    free(C2);
	
    return 0;
}
