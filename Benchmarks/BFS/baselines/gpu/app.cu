#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <stdint.h>

#include "../../support/common.h"
#include "../../support/graph.h"
#include "../../support/params.h"
#include "../../support/timer.h"
#include "../../support/utils.h"

__global__ void bfs_kernel(CSRGraph csrGraph, uint32_t* nodeLevel, uint32_t* prevFrontier, uint32_t* currFrontier, uint32_t numPrevFrontier, uint32_t* numCurrFrontier,  uint32_t level) {
    uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < numPrevFrontier) {
        uint32_t node = prevFrontier[i];
        for(uint32_t edge = csrGraph.nodePtrs[node]; edge < csrGraph.nodePtrs[node + 1]; ++edge) {
            uint32_t neighbor = csrGraph.neighborIdxs[edge];
            if(atomicCAS(&nodeLevel[neighbor], UINT32_MAX, level) == UINT32_MAX) { // Node not previously visited
                uint32_t currFrontierIdx = atomicAdd(numCurrFrontier, 1);
                currFrontier[currFrontierIdx] = neighbor;
            }
        }
    }
}

int main(int argc, char** argv) {

    // Process parameters
    struct Params p = input_params(argc, argv);

    // Initialize BFS data structures
    PRINT_INFO(p.verbosity >= 1, "Reading graph %s", p.fileName);
    struct COOGraph cooGraph = readCOOGraph(p.fileName);
    PRINT_INFO(p.verbosity >= 1, "    Graph has %d nodes and %d edges", cooGraph.numNodes, cooGraph.numEdges);
    struct CSRGraph csrGraph = coo2csr(cooGraph);
    uint32_t* nodeLevel_cpu = (uint32_t*) malloc(csrGraph.numNodes*sizeof(uint32_t));
    uint32_t* nodeLevel_gpu = (uint32_t*) malloc(csrGraph.numNodes*sizeof(uint32_t));
    for(uint32_t i = 0; i < csrGraph.numNodes; ++i) {
        nodeLevel_cpu[i] = UINT32_MAX; // Unreachable
        nodeLevel_gpu[i] = UINT32_MAX; // Unreachable
    }
    uint32_t srcNode = 0;

    // Allocate GPU memory
    CSRGraph csrGraph_d;
    csrGraph_d.numNodes = csrGraph.numNodes;
    csrGraph_d.numEdges = csrGraph.numEdges;
    hipMalloc((void**) &csrGraph_d.nodePtrs, (csrGraph_d.numNodes + 1)*sizeof(uint32_t));
    hipMalloc((void**) &csrGraph_d.neighborIdxs, csrGraph_d.numEdges*sizeof(uint32_t));
    uint32_t* nodeLevel_d;
    hipMalloc((void**) &nodeLevel_d, csrGraph_d.numNodes*sizeof(uint32_t));
    uint32_t* buffer1_d;
    hipMalloc((void**) &buffer1_d, csrGraph_d.numNodes*sizeof(uint32_t));
    uint32_t* buffer2_d;
    hipMalloc((void**) &buffer2_d, csrGraph_d.numNodes*sizeof(uint32_t));
    uint32_t* numCurrFrontier_d;
    hipMalloc((void**) &numCurrFrontier_d, sizeof(uint32_t));
    uint32_t* prevFrontier_d = buffer1_d;
    uint32_t* currFrontier_d = buffer2_d;

    // Copy data to GPU
    hipMemcpy(csrGraph_d.nodePtrs, csrGraph.nodePtrs, (csrGraph_d.numNodes + 1)*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(csrGraph_d.neighborIdxs, csrGraph.neighborIdxs, csrGraph_d.numEdges*sizeof(uint32_t), hipMemcpyHostToDevice);
    nodeLevel_gpu[srcNode] = 0;
    hipMemcpy(nodeLevel_d, nodeLevel_gpu, csrGraph_d.numNodes*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(prevFrontier_d, &srcNode, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Calculating result on GPU
    PRINT_INFO(p.verbosity >= 1, "Calculating result on GPU");
    Timer timer;
    startTimer(&timer);
    uint32_t numPrevFrontier = 1;
    uint32_t numThreadsPerBlock = 256;
    for(uint32_t level = 1; numPrevFrontier > 0; ++level) {

        // Visit nodes in previous frontier
        hipMemset(numCurrFrontier_d, 0, sizeof(uint32_t));
        uint32_t numBlocks = (numPrevFrontier + numThreadsPerBlock - 1)/numThreadsPerBlock;
        bfs_kernel <<< numBlocks, numThreadsPerBlock >>> (csrGraph_d, nodeLevel_d, prevFrontier_d, currFrontier_d, numPrevFrontier, numCurrFrontier_d, level);

        // Swap buffers
        uint32_t* tmp = prevFrontier_d;
        prevFrontier_d = currFrontier_d;
        currFrontier_d = tmp;
        hipMemcpy(&numPrevFrontier, numCurrFrontier_d, sizeof(uint32_t), hipMemcpyDeviceToHost);

    }
    hipDeviceSynchronize();
    stopTimer(&timer);
    if(p.verbosity == 0) PRINT("%f", getElapsedTime(timer)*1e3);
    PRINT_INFO(p.verbosity >= 1, "Elapsed time: %f ms", getElapsedTime(timer)*1e3);

    // Copy data from GPU
    hipMemcpy(nodeLevel_gpu, nodeLevel_d, csrGraph_d.numNodes*sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Initialize frontier double buffers for CPU
    uint32_t* buffer1 = (uint32_t*) malloc(csrGraph.numNodes*sizeof(uint32_t));
    uint32_t* buffer2 = (uint32_t*) malloc(csrGraph.numNodes*sizeof(uint32_t));
    uint32_t* prevFrontier = buffer1;
    uint32_t* currFrontier = buffer2;

    // Calculating result on CPU
    PRINT_INFO(p.verbosity >= 1, "Calculating result on CPU");
    nodeLevel_cpu[srcNode] = 0;
    prevFrontier[0] = srcNode;
    numPrevFrontier = 1;
    for(uint32_t level = 1; numPrevFrontier > 0; ++level) { 

        uint32_t numCurrFrontier = 0;

        // Visit nodes in the previous frontier
        for(uint32_t i = 0; i < numPrevFrontier; ++i) {
            uint32_t node = prevFrontier[i];
            for(uint32_t edge = csrGraph.nodePtrs[node]; edge < csrGraph.nodePtrs[node + 1]; ++edge) {
                uint32_t neighbor = csrGraph.neighborIdxs[edge];
                if(nodeLevel_cpu[neighbor] == UINT32_MAX) { // Node not previously visited
                    nodeLevel_cpu[neighbor] = level;
                    currFrontier[numCurrFrontier] = neighbor;
                    ++numCurrFrontier;
                }
            }
        }

        // Swap buffers
        uint32_t* tmp = prevFrontier;
        prevFrontier = currFrontier;
        currFrontier = tmp;
        numPrevFrontier = numCurrFrontier;

    }

    // Verify result
    PRINT_INFO(p.verbosity >= 1, "Verifying the result");
    for(uint32_t i = 0; i < csrGraph.numNodes; ++i) {
        if(nodeLevel_cpu[i] != nodeLevel_gpu[i]) {
            printf("Mismatch detected at node %u (CPU result = %u, GPU result = %u)\n", i, nodeLevel_cpu[i], nodeLevel_gpu[i]);
            exit(0);
        }
    }

    // Deallocate data structures
    freeCOOGraph(cooGraph);
    freeCSRGraph(csrGraph);
    free(nodeLevel_cpu);
    free(nodeLevel_gpu);
    free(buffer1);
    free(buffer2);

    return 0;

}

