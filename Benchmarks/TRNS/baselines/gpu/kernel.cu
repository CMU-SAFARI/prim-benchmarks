#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016 University of Cordoba and University of Illinois
 * All rights reserved.
 *
 * Developed by:    IMPACT Research Group
 *                  University of Cordoba and University of Illinois
 *                  http://impact.crhc.illinois.edu/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the 
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of IMPACT Research Group, University of Cordoba, 
 *        University of Illinois nor the names of its contributors may be used 
 *        to endorse or promote products derived from this Software without 
 *        specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 */

#include "support/common.h"

extern __shared__ int l_mem[];

// GPU kernel ------------------------------------------------------------------------------------------
__global__ void PTTWAC_soa_asta(int A, int B, int b, T *input, int *finished, int *head) {

    int* done = l_mem;
    int* gid_ = &done[1];

    const int tid = threadIdx.x;
    int       m   = A * B - 1;

    if(tid == 0) // Dynamic fetch
        gid_[0] = atomicAdd(&head[0], 1);
    __syncthreads();

    while(gid_[0] < m) {
        int next_in_cycle = (gid_[0] * A) - m * (gid_[0] / B);
        if(next_in_cycle == gid_[0]) {
            if(tid == 0) // Dynamic fetch
                gid_[0] = atomicAdd(&head[0], 1);
            __syncthreads();
            continue;
        }
        T   data1, data2, data3, data4;
        int i = tid;
        if(i < b)
            data1 = input[gid_[0] * b + i];
        i += blockDim.x;
        if(i < b)
            data2 = input[gid_[0] * b + i];
        i += blockDim.x;
        if(i < b)
            data3 = input[gid_[0] * b + i];
        i += blockDim.x;
        if(i < b)
            data4 = input[gid_[0] * b + i];

        if(tid == 0) {
            //make sure the read is not cached
            done[0] = atomicAdd(&finished[gid_[0]], 0);
        }
        __syncthreads();

        for(; done[0] == 0; next_in_cycle = (next_in_cycle * A) - m * (next_in_cycle / B)) {
            T backup1, backup2, backup3, backup4;
            i = tid;
            if(i < b)
                backup1 = input[next_in_cycle * b + i];
            i += blockDim.x;
            if(i < b)
                backup2 = input[next_in_cycle * b + i];
            i += blockDim.x;
            if(i < b)
                backup3 = input[next_in_cycle * b + i];
            i += blockDim.x;
            if(i < b)
                backup4 = input[next_in_cycle * b + i];

            if(tid == 0) {
                done[0] = atomicExch(&finished[next_in_cycle], (int)1);
            }
            __syncthreads();

            if(!done[0]) {
                i = tid;
                if(i < b)
                    input[next_in_cycle * b + i] = data1;
                i += blockDim.x;
                if(i < b)
                    input[next_in_cycle * b + i] = data2;
                i += blockDim.x;
                if(i < b)
                    input[next_in_cycle * b + i] = data3;
                i += blockDim.x;
                if(i < b)
                    input[next_in_cycle * b + i] = data4;
            }
            i = tid;
            if(i < b)
                data1 = backup1;
            i += blockDim.x;
            if(i < b)
                data2 = backup2;
            i += blockDim.x;
            if(i < b)
                data3 = backup3;
            i += blockDim.x;
            if(i < b)
                data4 = backup4;
        }

        if(tid == 0) // Dynamic fetch
            gid_[0] = atomicAdd(&head[0], 1);
        __syncthreads();
    }
}

hipError_t call_PTTWAC_soa_asta(int blocks, int threads, int A, int B, int b, T *input, 
    int *finished, int *head, int l_mem_size){
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    PTTWAC_soa_asta<<<dimGrid, dimBlock, l_mem_size>>>(A, B, b, input, 
        finished, head);
    hipError_t err = hipGetLastError();
    return err;
}

__global__ void BS_marshal(T *input, int tile_size, int width) {

  T* tile = (T*)l_mem;

  int tidx = threadIdx.x;
  int m = width*tile_size-1;
  int bid = blockIdx.x;

  input += tile_size*width*bid;
  for (int i = tidx; i < tile_size*width; i+=blockDim.x) {
    int next = (i * tile_size)-m*(i/width);
    tile[next] = input[i];
  }
  __syncthreads();
  for (int i = tidx; i < tile_size*width; i+=blockDim.x) {
    input[i] = tile[i];
  }
}

hipError_t call_BS_marshal(int blocks, int threads, int m, int n, T *input, int l_mem_size){
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    BS_marshal<<<dimGrid, dimBlock, l_mem_size>>>(input, m, n);
    hipError_t err = hipGetLastError();
    return err;
}
