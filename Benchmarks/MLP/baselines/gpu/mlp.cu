#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "../../support/common.h"

#define THREAD 128

__global__ void gemv(int m, int n, T *adim, T *b, T *d_ans);

void cgemv(int m, int n, T *adim, T *b, T *d_ans);

double gettime()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + (double)tv.tv_usec*1.0e-6;
}

int main(int argc, char **argv)
{
	/* for CPU */
	int i, j;
	T **bdim; 
	T *c, *ans, *h_ans, *h_c;
	int n = 8192;
	int m = 20480;

	bdim = (T**) malloc(NUM_LAYERS * sizeof(T*));
	for(int l = 0; l < NUM_LAYERS; l++)
		bdim[l] = (T*)malloc(sizeof(T)*m*n);
	c = (T*)malloc(sizeof(T) *n);
	h_c = (T*)malloc(sizeof(T) *n);
	ans = (T*)malloc(sizeof(T) *m);
	h_ans = (T*)malloc(sizeof(T) *m);

	/* for GPU */
	T *d_bdim; 
	T *d_c, *d_ans;
	hipMalloc((void **)&d_bdim, sizeof(T)*m*n);
	hipMalloc((void **)&d_c, sizeof(T)*n);
	hipMalloc((void **)&d_ans, sizeof(T)*m);

	for(i = 0; i < n; i++)
	{
		if(i % 50 < 48)
		{
			c[i] = 0;
			h_c[i] = 0;
		}
		else
		{
			c[i] = i % 2;
			h_c[i] = i % 2;
		}
	}
	for(int l = 0; l < NUM_LAYERS; l++)
		for(i = 0; i < n; i++)
		{
			for(j = 0; j < m; j++){
				if(j % 100 < 98)
				{

					bdim[l][i*m+j] = 0;
				}
				else
				{

					bdim[l][i*m+j] = (l + i) % 2;
				}
			}
		}

	for(j = 0; j < m; j++){
		ans[j] = 0;
		h_ans[j] = 0;
	}
	// Computation on the host for verification
	T* vector = c;
	T* output = ans;
	T* matrix;
	int mm = m;
	int nn = n;
	for(int l = 0; l < NUM_LAYERS; l++){
		matrix = bdim[l];
		cgemv(mm, nn, matrix, vector, output);
		vector = output;
                h_ans = output;
		mm = n; nn = m;
	}

	// Event creation
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time1 = 0;
	float time2 = 0;
	hipMemcpy(d_ans, h_ans, sizeof(T)*m, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, sizeof(T)*n, hipMemcpyHostToDevice);

	vector = d_c;
	output = d_ans;
	mm = m;
	nn = n;
	for(int l = 0; l < NUM_LAYERS; l++){
		hipMemcpy(d_bdim, bdim[l], sizeof(T)*m*n, hipMemcpyHostToDevice);
		matrix = d_bdim;
		// Start timer
		hipEventRecord( start, 0 );
		gemv<<<mm, THREAD>>>(mm, nn, matrix, vector, output);
		// End timer
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &time2, start, stop );
		time1 += time2;
		vector = output;
		d_ans = output;
		mm = n; nn = m;
	}

	hipMemcpy(h_ans, d_ans, sizeof(T)*m, hipMemcpyDeviceToHost);
	hipMemcpy(h_c, d_c, sizeof(T)*n, hipMemcpyDeviceToHost);

	for(i = 0; i < m; i++)
	{
		if(ans[i] != h_ans[i])
		printf("ERROR in Ans %d -> %d -- %d\n", i, ans[i], h_ans[i]);
        }

	for(i = 0; i < n; i++)
	{
		if(c[i] != h_c[i])
		printf("ERROR in C %d -> %d -- %d\n", i, c[i], h_c[i]);
	}
	printf("Execution time = %f ms\n", time1);


	for(int l = 0; l < NUM_LAYERS; l++)
		free(bdim[l]);


	free(bdim);
	free(c);
	free(ans);
	free(h_c);
	hipFree(d_bdim);
	hipFree(d_c);
	hipFree(d_ans);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
} 

__global__ void gemv(int m, int n, T* adim, T* b, T* d_ans)
{
	int i;
	int div = n/THREAD;
	__shared__ T tmp[THREAD];

	tmp[threadIdx.x] = 0.0;

	for(i = 0; i < div; i++){
		tmp[threadIdx.x] += adim[blockIdx.x*n+i*THREAD+threadIdx.x] * b[i * THREAD + threadIdx.x];
	}
	if(threadIdx.x < m%THREAD)
		tmp[threadIdx.x] += adim[blockIdx.x*n+THREAD*div+threadIdx.x] * b[THREAD * div + threadIdx.x];

	__syncthreads();

	for(i = THREAD / 2; i > 31; i = i / 2)
	{
		if(threadIdx.x < i)
			tmp[threadIdx.x] += tmp[threadIdx.x + i];
		__syncthreads();
	}

	if(threadIdx.x < 16)
	{
		tmp[threadIdx.x] += tmp[threadIdx.x + 16];
		__syncthreads();
		tmp[threadIdx.x] += tmp[threadIdx.x + 8];
		__syncthreads();
		tmp[threadIdx.x] += tmp[threadIdx.x + 4];
		__syncthreads();
		tmp[threadIdx.x] += tmp[threadIdx.x + 2];
		__syncthreads();
		tmp[threadIdx.x] += tmp[threadIdx.x + 1];
		__syncthreads();
	}


	if(threadIdx.x == 0)
		d_ans[blockIdx.x] = max(0, tmp[0]);

}

void cgemv(int m, int n, T *adim, T *b, T *d_ans)
{
	int i, j;

	for(i = 0; i < m; i++){
		for(j = 0; j < n; j++)
			d_ans[i] += adim[i*n+j] * b[j];
		d_ans[i] = max(0, d_ans[i]);
	}

}
