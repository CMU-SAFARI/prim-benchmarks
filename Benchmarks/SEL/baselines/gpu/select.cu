/***************************************************************************
 *cr
 *cr            (C) Copyright 2015 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/*
  In-Place Data Sliding Algorithms for Many-Core Architectures, presented in ICPP’15

  Copyright (c) 2015 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Authors: Juan Gómez-Luna (el1goluj@uco.es, gomezlun@illinois.edu), Li-Wen Chang (lchang20@illinois.edu)
*/

#include "ds.h"

// Sample predicate for partition (only for INT)
struct is_even{
  __host__ __device__
  bool operator()(const T &x){
    return (x % 2) == 0;
  }
};

#include "kernel.cu"

// Sequential CPU version
void cpu_copy_if(T* output, T* input, int elements, struct is_even pred){
  int pos = 0;
  for (int i = 0; i < elements; i++){
    if(pred(input[i])){
	  output[pos] = input[i];
      pos++;
    }
  }
}
void cpu_remove_if(T* input, int elements, struct is_even pred){
  int pos = 0;
  for (int i = 0; i < elements; i++){
    if(!pred(input[i])){
	  input[pos] = input[i];
      pos++;
    }
  }
}

int main(int argc, char **argv){

  // Syntax verification
  if (argc != 4) {
      printf("Wrong format\n");
      printf("Syntax: %s <Device Input (%% elements) numElements>\n",argv[0]);
      exit(1);
  }
  int device = atoi(argv[1]);
  int input = atoi(argv[2]);
  int numElements = atoi(argv[3]);
  size_t size = numElements * sizeof(T);

  // Set device
  hipDeviceProp_t device_properties;
  hipGetDeviceProperties(&device_properties,device);
  hipSetDevice(device);

  printf("DS Select on %s\n", device_properties.name);
  printf("Thread block size = %d\n", L_DIM);
  printf("Coarsening factor = %d\n", REGS);
#ifdef FLOAT
  printf("Single precision array: %d elements\n", numElements);
#elif INT
  printf("Integer array: %d elements\n", numElements);
#else
  printf("Double precision array: %d elements\n", numElements);
#endif

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float time1 = 0;
  float time2 = 0;
  float time3 = 0;

  // Allocate the host input vector A
  T *h_A = (T*)malloc(size);

  // Allocate the host output vectors
  T *h_B = (T*)malloc(size);
  T *h_C = (T*)malloc(size);
  T *h_D = (T*)malloc(size);

  // Allocate the device input vector A and output vector B
  T *d_A = NULL;
  hipMalloc((void **)&d_A, size);
  T *d_B = NULL;
  hipMalloc((void **)&d_B, size);

#define WARMUP 2
#define REP 10
  unsigned int flagM1 = 0;
  unsigned int flagM2 = 0;
  for(int iteration = 0; iteration < REP+WARMUP; iteration++){
    // Initialize the host input vectors
    srand(2014);
    for(int i = 0; i < numElements; i++)
        h_A[i] = i % 2 != 0 ? i:i+1;
    int M = (numElements * input)/100;
    int m = M;
    while(m>0){
        int x = (int)(numElements*(((float)rand()/(float)RAND_MAX)));
        if(h_A[x] % 2 != 0){
            h_A[x] = x * 2;
            m--;
        }
    }

#if PRINT
    for(int i = 0; i < numElements; ++i){
        printf("%d ",*(h_A+i));
    }
    printf("\n");
#endif

    // Copy the host input vector A in host memory to the device input vector in device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int ldim = L_DIM;
    // Atomic flags
    unsigned int* d_flags = NULL;
    const int num_flags = numElements % (ldim * REGS) == 0 ? numElements / (ldim * REGS) : numElements / (ldim * REGS) + 1;
    unsigned int *flags = (unsigned int *)calloc(sizeof(unsigned int), num_flags + 2);
    flags[0] = 1;
    flags[num_flags + 1] = 0;
    hipMalloc((void **)&d_flags, (num_flags + 2) * sizeof(unsigned int));
    hipMemcpy(d_flags, flags, (num_flags + 2) * sizeof(unsigned int), hipMemcpyHostToDevice);
    // Number of work-groups/thread blocks
    int num_wg = num_flags;

    // Start timer
    hipEventRecord( start, 0 );

    // Kernel launch (Copy_if)
    select_copy_if<<<num_wg, ldim>>>(d_B, d_A, numElements, d_flags, is_even());

    hipMemcpy(&flagM1, d_flags + num_flags, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Stop timer
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time1, start, stop );
    if(iteration >= WARMUP) time2 += time1;

    if(iteration == REP+WARMUP-1){
      float timer = time2 / REP;
      double bw = (double)((numElements + flagM1) * sizeof(T)) / (double)(timer * 1000000.0);
      printf("Copy_if - Execution time = %f ms, Throughput = %f GB/s\n", timer, bw);
    }

    // Atomic flags
    hipMemcpy(d_flags, flags, (num_flags + 2) * sizeof(unsigned int), hipMemcpyHostToDevice);
    free(flags);

    // Start timer
    hipEventRecord( start, 0 );

    // Kernel launch (Remove_if)
    select_remove_if<<<num_wg, ldim>>>(d_A, d_A, numElements, d_flags, is_even()); 

    hipMemcpy(&flagM2, d_flags + num_flags, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // End timer
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time1, start, stop );
    if(iteration >= WARMUP) time3 += time1;

    if(iteration == REP+WARMUP-1){
      float timer = time3 / REP;
      double bw = (double)((numElements + flagM2) * sizeof(T)) / (double)(timer * 1000000.0);
      printf("Remove_if - Execution time = %f ms, Throughput = %f GB/s\n", timer, bw);
    }

    // Free flags
    hipFree(d_flags);
  }
  // Copy to host memory
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_A, size, hipMemcpyDeviceToHost);

  // CPU execution for comparison
  cpu_copy_if(h_D, h_A, numElements, is_even());
  cpu_remove_if(h_A, numElements, is_even());

  // Verify that the result vector is correct
#if PRINT
  for(int i = 0; i < numElements; ++i){
     printf("%d ",*(h_B+i));
  }
  printf("\n");
  for(int i = 0; i < numElements; ++i){
      printf("%d ",*(h_D+i));
  }
  printf("\n");
#endif
  for (int i = 0; i < flagM1 - 1; ++i){
      if (h_B[i] != h_D[i]){
          fprintf(stderr, "Copy_if - Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
      }
  }
  for (int i = 0; i < flagM2 - 1; ++i){
      if (h_C[i] != h_A[i]){
          fprintf(stderr, "Remove_if - Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
      }
  }
  printf("Test PASSED\n");

  // Free device global memory
  hipFree(d_A);
  hipFree(d_B);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_D);

  return 0;
}
